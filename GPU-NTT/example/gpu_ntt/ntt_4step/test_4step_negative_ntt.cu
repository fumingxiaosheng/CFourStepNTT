#include <cstdlib>
#include <random>

#include "ntt.cuh"
#include "ntt_4step.cuh"
#include "ntt_4step_cpu.cuh"
//#include "hxw_constant.cuh"

#define DEFAULT_MODULUS

using namespace std;

int LOGN;
int BATCH;
int N;


extern __constant__ Root Csitable64[32];

int main(int argc, char* argv[])
{
    printf("hxw negative ntt test\n");
    
    CudaDevice();

    //根据传入的参数指定LOGN和BATCH
    if(argc < 3)
    {
        LOGN = 21;
        BATCH = 1;
    }
    else
    {
        LOGN = atoi(argv[1]);
        BATCH = atoi(argv[2]);
    }

//定义约减的方式
#ifdef BARRETT_64
    ModularReductionType modular_reduction_type = ModularReductionType::BARRET;
#elif defined(GOLDILOCKS_64)
    ModularReductionType modular_reduction_type = ModularReductionType::GOLDILOCK;
#elif defined(PLANTARD_64)
    ModularReductionType modular_reduction_type = ModularReductionType::PLANTARD;
#else
#error "Please define reduction type."
#endif

    // Current 4step NTT implementation only works for ReductionPolynomial::X_N_minus!
    NTTParameters4Step parameters(LOGN, modular_reduction_type, ReductionPolynomial::X_N_minus);
    
    // NTT generator with certain modulus and root of unity
    NTT_4STEP_CPU generator(parameters);

    std::random_device rd;
    std::mt19937 gen(rd());
    unsigned long long minNumber = 0;
    unsigned long long maxNumber = parameters.modulus.value - 1;
    std::uniform_int_distribution<unsigned long long> dis(minNumber, maxNumber);

    // Random data generation for polynomials
    vector<vector<Data>> input1(BATCH);
    for(int j = 0; j < BATCH; j++)
    {
        for(int i = 0; i < parameters.n; i++)
        {
            input1[j].push_back(dis(gen));
        }
    }

    // Performing CPU NTT
    vector<vector<Data>> ntt_result(BATCH);
    for(int i = 0; i < BATCH; i++)
    {
        ntt_result[i] = generator.negative_ntt(input1[i]);
    }
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////

    Data* Input_Datas;

    THROW_IF_CUDA_ERROR(hipMalloc(&Input_Datas, BATCH * parameters.n * sizeof(Data)));

    Data* Output_Datas;
    THROW_IF_CUDA_ERROR(hipMalloc(&Output_Datas, BATCH * parameters.n * sizeof(Data)));

    
    for(int j = 0; j < BATCH; j++)
    {
        //input1[j].data()代表指向vector的第一个数据的指针
        THROW_IF_CUDA_ERROR(hipMemcpy(Input_Datas + (parameters.n * j), input1[j].data(),
                                       parameters.n * sizeof(Data), hipMemcpyHostToDevice));

        THROW_IF_CUDA_ERROR(hipMemcpy(Output_Datas + (parameters.n * j), input1[j].data(),
                                       parameters.n * sizeof(Data), hipMemcpyHostToDevice));
    }

    //////////////////////////////////////////////////////////////////////////

    //传输相应的旋转因子表

    vector<Root_> psitable1 =
        parameters.gpu_root_of_unity_table_generator(parameters.negative_2n1_based_root_of_unity_table);//ROOT_是根据约减类型而定的数据类型，注意在param宏宏,root是data类型的
    Root* psitable_device1; //Root是根据具体的约减类型确定的,其和Root_是相同的
    THROW_IF_CUDA_ERROR(hipMalloc(&psitable_device1, parameters.n1* sizeof(Root)));
    THROW_IF_CUDA_ERROR(hipMemcpy(psitable_device1, psitable1.data(),
                                   parameters.n1 * sizeof(Root), hipMemcpyHostToDevice));

    vector<Root_> psitable2 =
        parameters.gpu_root_of_unity_table_generator(parameters.negative_n2_based_root_of_unity_table);
    Root* psitable_device2;
    THROW_IF_CUDA_ERROR(hipMalloc(&psitable_device2, (parameters.n2 >> 1) * sizeof(Root)));
    THROW_IF_CUDA_ERROR(hipMemcpy(psitable_device2, psitable2.data(),
                                   (parameters.n2 >> 1) * sizeof(Root), hipMemcpyHostToDevice));

    Root* W_Table_device;
    THROW_IF_CUDA_ERROR(hipMalloc(&W_Table_device, parameters.n * sizeof(Root)));
    THROW_IF_CUDA_ERROR(hipMemcpy(W_Table_device, parameters.negative_W_root_of_unity_table.data(),
                                   parameters.n * sizeof(Root), hipMemcpyHostToDevice));

    vector<Root_> psitable32 =
        parameters.gpu_root_of_unity_table_generator(parameters.n32_root_of_unity_table);
    Root* psitable_device32;

    THROW_IF_CUDA_ERROR(hipMalloc(&psitable_device32, 16 * sizeof(Root)));
    THROW_IF_CUDA_ERROR(hipMemcpy(psitable_device32, psitable32.data(), 16 * sizeof(Root), hipMemcpyHostToDevice));

    Root* n32_W_Table_device;
    THROW_IF_CUDA_ERROR(hipMalloc(&n32_W_Table_device, 1024 * sizeof(Root)));
    THROW_IF_CUDA_ERROR(hipMemcpy(n32_W_Table_device, parameters.n32_W_root_of_unity_table.data(), 1024 * sizeof(Root), hipMemcpyHostToDevice));

    //////////////////////////////////////////////////////////////////////////

    Modulus* test_modulus;//用于存放模数
    THROW_IF_CUDA_ERROR(hipMalloc(&test_modulus, sizeof(Modulus)));

    Modulus test_modulus_[1] = {parameters.modulus};

    THROW_IF_CUDA_ERROR(
        hipMemcpy(test_modulus, test_modulus_, sizeof(Modulus), hipMemcpyHostToDevice));

    Ninverse* test_ninverse;//n^-1在Zq上的值,device变量指针
    THROW_IF_CUDA_ERROR(hipMalloc(&test_ninverse, sizeof(Ninverse)));

    Ninverse test_ninverse_[1] = {parameters.n_inv};

    THROW_IF_CUDA_ERROR(
        hipMemcpy(test_ninverse, test_ninverse_, sizeof(Ninverse), hipMemcpyHostToDevice));

    ntt4step_rns_configuration cfg_init = {.n_power = LOGN,
                                      .ntt_type = FORWARD,
                                      .mod_inverse = test_ninverse,
                                      .stream = 0};

    
    GPU_NEGATIVE_4STEP_NTT(Output_Datas, Input_Datas, psitable_device1, psitable_device2, W_Table_device, psitable_device32, n32_W_Table_device, test_modulus, cfg_init, BATCH, 1);

    vector<Data> Output_Host(parameters.n * BATCH);
    hipMemcpy(Output_Host.data(), Input_Datas, parameters.n * BATCH * sizeof(Data),
               hipMemcpyDeviceToHost);


    hipDeviceSynchronize();
    // Comparing GPU NTT results and CPU NTT results
    //读出相应的数
    bool check = true;

    for(int i = 0; i < BATCH; i++)
    {
        check = check_result(Output_Host.data() + (i * parameters.n), ntt_result[i].data(),
                             parameters.n);

        if(!check)
        {
            cout << "(in " << i << ". Poly.)" << endl;
            break;
        }

        if((i == (BATCH - 1)) && check)
        {
            cout << "origaninal All Correct." << endl;
        }
    }

    return EXIT_SUCCESS;
}